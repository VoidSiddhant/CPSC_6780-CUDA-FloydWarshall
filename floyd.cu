#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>
#include <cassert>
#include <sys/time.h>

using namespace std;
using namespace std::chrono;


// Kernel function for inner two loop of Floyd Warshall Algorithm
__global__ void GPUInnerLoops(int V, int k, int* dis)
{
	int tid = (blockDim.x * blockDim.y) * threadIdx.z + (threadIdx.y * blockDim.x) + (threadIdx.x);
	int bid = (gridDim.x * gridDim.y) * blockIdx.z + (blockIdx.y * gridDim.x) + (blockIdx.x);
	int T = blockDim.x * blockDim.y * blockDim.z;
	int B = gridDim.x * gridDim.y * gridDim.z;

	int tm = 0;
    
	/*
	 * Each cell in the matrix is assigned to a different thread. 
	 * Each thread do O(N*number of asssigned cell) computation.
	 * Assigned cells of different threads does not overlape with
	 * each other. And so no need for synchronization.
	 */
     
	for (int i = bid; i < V; i += B)
	{
		for (int j = tid; j < V; j += T)
		{
			tm = dis[i * V + k] + dis[k * V + j];
			dis[i * V + j] = tm * (tm < dis[i * V + j]) + dis[i * V + j] * (tm >= dis[i * V + j]);
		}
	}
}

const int V = 1000;

const int INF = 99999;


void FloydWarshall(int* dis)
{

	// Timer functions
    float timeElapsed = 0.0f;
	hipEvent_t timeStart, timeStop;
	hipEventCreate(&timeStart);
	hipEventCreate(&timeStop);
	hipEventRecord(timeStart, 0);
    
    
	for (int k = 0; k < V; k++)
	{
		GPUInnerLoops << <dim3(16, 8, 8), dim3(8, 8, 8) >> > (V, k, dis);
		hipDeviceSynchronize();
	}
    

	hipEventRecord(timeStop, 0);
	hipEventSynchronize(timeStop);
	hipEventElapsedTime(&timeElapsed, timeStart, timeStop);
	std::cout << "GPU-Floyd Execution took : " << timeElapsed/1000.0f << " seconds" << endl;
	hipEventDestroy(timeStart);
	hipEventDestroy(timeStop);
}

int main(void)
{

	int* dis;

	// Allocate Unified Memory - accessible from CPU or GPU
	hipMallocManaged(&dis, V * V * sizeof(int));

	//initialize dis array on the host
	for (int i = 0; i < V; i++)
	{
		for (int j = 0; j < V; j++)
		{

			if (j == i + 1) dis[i * V + j] = 1;
			else if (i != j) dis[i * V + j] = INF;
			else dis[i * V + j] = 0;
		}
	}
	FloydWarshall(dis);

	for (int i = 0; i < V; i++)
	{
		for (int j = 0; j < V; j++)
		{
			if (j >= i)
			{
				assert(dis[i * V + j] == j - i);
			}
			else assert(dis[i * V + j] == INF);
		}
	}


	// Free memory
	hipFree(dis);
	return 0;
}

